#include "hip/hip_runtime.h"
#include <stdio.h>
#include <getopt.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include "timer.h"
#include <sys/stat.h>
#include <errno.h>
#include <string.h>

#define N 8192
//#define M 16

bool print_matrix = false;
bool validation = false;

float a[N][N], b[N][N], c[N][N];

__global__ void cu_mat_mul(float *A, float *B, float *C, int n)
{
	int i, j ,k;
	float acm = 0;
	i = blockDim.x * blockIdx.x + threadIdx.x;
	j = blockDim.y * blockIdx.y + threadIdx.y;

	//printf("i - %d, j - %d\n", i, j);

	for(k = 0; k < n ; k++)
	{
		acm += *( A + j*n + k ) * *(B + k*n + i);
	}

	*(C + j*n + i) = acm;
}

void mat_mul() {

	int sizeA, sizeB, sizeC;
	float *d_a, *d_b, *d_c;

	sizeA = sizeB = sizeC = N*N;

	if(hipMalloc(&d_a, sizeof(float)*sizeA) != hipSuccess)
	{
		printf("Malloc A error!\n");
		exit(1);
	}

	if(hipMalloc(&d_b, sizeof(float)*sizeB) != hipSuccess)
	{
		printf("Malloc B error!\n");
		exit(1);
	}

	if(hipMalloc(&d_c, sizeof(float)*sizeC) != hipSuccess)
	{
		printf("Malloc C error!\n");
		exit(1);
	}

	if( hipMemcpy(d_a, a, sizeA*4, hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Memcpy A error!\n");
		exit(1);
	}
	
	if( hipMemcpy(d_b, b, sizeB*4, hipMemcpyHostToDevice) != hipSuccess)
	{
		printf("Memcpy B error!\n");
		exit(1);
	}

	dim3 grid( 16, 16, 1 );
	dim3 thread( N/grid.x, N/grid.y, 1 );

	cu_mat_mul <<< thread, grid >>> (d_a, d_b, d_c, N);

	if( hipMemcpy(c, d_c, sizeof(float)*sizeC, hipMemcpyDeviceToHost) != hipSuccess)
	{
		printf("Memcpy C error!\n");
		exit(1);
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return;
}
/*
 * ==================================================================
 *                      DO NOT EDIT BELOW THIS LINE
 * ==================================================================
 */

void check_mat_mul() {
    printf("Validating...\n");

    bool is_valid = true;
    float eps = 1e-3;
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            float s = 0;
            for (int k = 0; k < N; ++k) {
                s += a[i][k] * b[k][j];
            }
            if (fabsf(c[i][j] - s) > eps && (s == 0 || fabsf((c[i][j] - s) / s) > eps)) {
                printf("c[%d][%d] : correct_value = %f, your_value = %f\n", i, j, s, c[i][j]);
                is_valid = false;
            }
        }
    }

    if (is_valid) {
        printf("result: VALID\n");
    } else {
        printf("result: INVALID\n");
    }
}

void generate_mat(float (*a)[N]) {
    for (int i = 0; i < N; ++i) { 
        for (int j = 0; j < N; ++j) {
            a[i][j] = (float)rand() / RAND_MAX - 0.5;
        }
    }
}

void print_mat(float (*a)[N]) {
    for (int i = 0; i < N; ++i) { 
        for (int j = 0; j < N; ++j) {
            printf("%+.3f ", a[i][j]);
        }
        printf("\n");
    }
}

void print_help(const char* prog_name) {
    printf("Usage: %s [-pvh]\n", prog_name);
    printf("OPTIONS\n");
    printf("  -p : print matrix data.\n");
    printf("  -v : validate matrix multiplication.\n");
    printf("  -h : print this page.\n");
}

void parse_opt(int argc, char **argv) {
    int opt;
    while ((opt = getopt(argc, argv, "pvh")) != -1) {
        switch(opt) {
            case 'p':
                print_matrix = true;
                break;
            case 'v':
                validation = true;
                break;
            case 'h':
            default:
                print_help(argv[0]);
                exit(0);
        }
    }
}

int main(int argc, char **argv) {
    parse_opt( argc, argv );

    generate_mat(a);
    generate_mat(b);

    printf("Calculating..."); fflush(stdout);
    timer_start(0);
    mat_mul();
    double elapsed_time = timer_stop(0);
    printf(" done!\n");

    if (print_matrix) {
        printf("MATRIX A:\n"); print_mat(a);
        printf("MATRIX B:\n"); print_mat(b);
        printf("MATRIX C:\n"); print_mat(c);
    }

    if (validation) {
        check_mat_mul();
    } else {
        printf("Validation is skipped.\n");
    }

    printf("Elapsed time: %f sec\n", elapsed_time);

    return 0;
}
